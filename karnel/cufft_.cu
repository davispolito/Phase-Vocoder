#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cufft_.h"
#include <chrono>
#include <hipfft/hipfft.h>

namespace FFT {
	namespace CuFFT {
		using FFT::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		/**
		 *Computes FFT using cuda library
		 */
		float2* computeCuFFT(float2* h_signal, int size) {
			float2 *d_signal;
			
			hipMalloc((void**)&d_signal, sizeof(float2*) * size);
			hipMemcpy(d_signal, h_signal, sizeof(float2*) * size, hipMemcpyHostToDevice);
			timer().startGpuTimer();
			hipfftHandle plan; 
			hipfftPlan1d(&plan, size * sizeof(float2*), HIPFFT_C2C, 1);

			hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
			timer().endGpuTimer();
			float2 *o_signal;
			o_signal = (float2*) malloc(size * sizeof(float2));
			hipMemcpy(o_signal, d_signal, sizeof(float2) * size, hipMemcpyDeviceToHost);

			hipFree(d_signal);
			return o_signal;
		}
	}
}
