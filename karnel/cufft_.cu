#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cufft_.h"
#include <chrono>
#include <hipfft/hipfft.h>

namespace FFT {
	namespace CuFFT {
		using FFT::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		/**
		 *Computes FFT using cuda library
		 */
		void computeCuFFT(float2* h_signal, int size) {
			timer().startGpuTimer();
			hipStreamAttachMemAsync(NULL, h_signal, 0, hipMemAttachGlobal);
			hipfftHandle plan; 
			hipfftPlan1d(&plan, size * sizeof(float2*), HIPFFT_C2C, 1);
			hipfftExecC2C(plan, (hipfftComplex *)h_signal, (hipfftComplex *)h_signal, HIPFFT_FORWARD);
			timer().endGpuTimer();
		}
	}
}
