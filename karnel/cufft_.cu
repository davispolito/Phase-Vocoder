#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cufft_.h"
#include <chrono>
#include "hipfft/hipfft.h"

namespace FFT {
	namespace CuFFT {
		using FFT::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		/**
		 *Computes FFT using cuda library
		 */
		void computeCuFFT(float2* h_signal, int size) {
			float2* d_signal;
			hipMalloc((void**)d_signal, sizeof(float2*) * size);
			hipMemcpy(d_signal, h_signal, sizeof(float2*) * size, hipMemcpyHostToDevice);
			timer().startGpuTimer();
			hipfftHandle plan; 
			hipfftPlan1d(&plan, size * sizeof(float2*), HIPFFT_C2C, 1);

			hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
			timer().endGpuTimer();
			hipMemcpy(h_signal, d_signal, sizeof(float2*) * size, hipMemcpyDeviceToHost);

			free(d_signal);
		}
	}
}
